#include "hip/hip_runtime.h"
#include "RubiksCube.h"
#include <string>
#include <iostream>

RubiksCube::RubiksCube(int size) {
	this->size = size;
	this->data = new unsigned char[size * size * 6];
	if (size == 2) {
		//first face
		condition.push_back(0);
		condition.push_back(0);
		condition.push_back(1);
		condition.push_back(0);
		condition.push_back(2);
		condition.push_back(0);
		condition.push_back(3);
		condition.push_back(0);
		//up edge
		condition.push_back(10);
		condition.push_back(2);
		condition.push_back(11);
		condition.push_back(2);
		//left edge
		condition.push_back(17);
		condition.push_back(4);
		condition.push_back(19);
		condition.push_back(4);
		//down edge
		condition.push_back(20);
		condition.push_back(5);
		condition.push_back(21);
		condition.push_back(5);
		//right edge
		condition.push_back(4);
		condition.push_back(1);
		condition.push_back(6);
		condition.push_back(1);
		
		//up back edge
		condition.push_back(8);
		condition.push_back(2);
		condition.push_back(9);
		condition.push_back(2);
		//left back edge
		condition.push_back(16);
		condition.push_back(4);
		condition.push_back(18);
		condition.push_back(4);
		//down back edge
		condition.push_back(22);
		condition.push_back(5);
		condition.push_back(23);
		condition.push_back(5);
		//right back edge
		condition.push_back(5);
		condition.push_back(1);
		condition.push_back(7);
		condition.push_back(1);
		//back face
		condition.push_back(12);
		condition.push_back(3);
		condition.push_back(13);
		condition.push_back(3);
		condition.push_back(14);
		condition.push_back(3);
		condition.push_back(15);
		condition.push_back(3);
	}
	if (false) {
		int RULDIndex[4] = { 13,22,40,49 };
		int RULDColor[4] = { 1,2,4,5 };
		int crossIndex[4] = { -3,1,3,-1 };
		int cornerIndex[4] = { -4,-2,4,2 };
		int edgeLeft[3] = { -4,-1,2 };
		int edgeRight[3] = { -2,1,4 };
		int edgeUp[3] = { -4,-3,-2 };
		int edgeDown[3] = { 2,3,4 };
		int UDL [3] = { -3,0,3 };
		int RLL [3] = { -1,0,1 };
		int faceIndex = 4;
		int faceColor = 0;
		//F center
		condition.push_back(faceIndex);
		condition.push_back(0);
		//F cross
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
		}
		//F corner
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
		}
		//R edge
		faceColor = 1;
		faceIndex = 13;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeLeft[i], faceColor);
		}
		//U edge
		faceColor = 2;
		faceIndex = 22;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeDown[i], faceColor);
		}
		//L edge
		faceColor = 4;
		faceIndex = 40;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeRight[i], faceColor);
		}
		//D edge
		faceColor = 5;
		faceIndex = 49;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeUp[i], faceColor);
		}
		//ring center
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, RULDIndex[i], RULDColor[i]);
		}
		//R middle
		faceColor = 1;
		faceIndex = 13;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + UDL[i], faceColor);
		}
		//U middle
		faceColor = 2;
		faceIndex = 22;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + RLL[i], faceColor);
		}
		//L middle
		faceColor = 4;
		faceIndex = 40;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + UDL[i], faceColor);
		}
		//D middle
		faceColor = 5;
		faceIndex = 49;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + RLL[i], faceColor);
		}
	}
	if (size == 3) {
		int RULDIndex[4] = { 13,22,40,49 };
		int RULDColor[4] = { 1,2,4,5 };
		int RULD2F[4] = { -1,3,1,-3 };
		int RULD2B[4] = { 1,-3,-1,3 };
		int RULD2FRC[4] = { 2,4,-2,-4 };
		int RULD2FLC[4] = { -4,2,4,-2 };
		int RULD2MLE[4] = {-3,-1,3,1};
		int RULD2MRE[4] = { 3,1,-3,-1 };
		int RULD2BRC[4] = {4,-2,-4,2};
		int RULD2BLC[4] = {-2,-4,2,4};

		int crossIndex[4] = { 1,-3,-1,3};
		int cornerIndex[4] = { -2,-4,2,4 };
		int edgeLeft[3] = { -4,-1,2 };
		int edgeRight[3] = { -2,1,4 };
		int edgeUp[3] = { -4,-3,-2 };
		int edgeDown[3] = { 2,3,4 };
		int UDL[3] = { -3,0,3 };
		int RLL[3] = { -1,0,1 };
		int faceIndex = 4;
		int faceColor = 0;
		//F center
		condition.push_back(faceIndex);
		condition.push_back(0);
		//F cross
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
			ConditionPush(condition, RULDIndex[i]+ RULD2F[i], RULDColor[i]);
		}
		//F conor
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
			ConditionPush(condition, RULDIndex[i] + RULD2FLC[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2FRC[plusIndex], RULDColor[plusIndex]);
		}
		//middle conor
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, RULDIndex[i] + RULD2MLE[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2MRE[plusIndex], RULDColor[plusIndex]);
		}
		//B cross Face
		faceIndex = 31;
		faceColor = 3;
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
		}
		//B corner Face
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
		}
		//RULD B Conr Face
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, RULDIndex[i] + RULD2BLC[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2BRC[plusIndex], RULDColor[plusIndex]);
		}
		//RULD B Cross Face
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, RULDIndex[i] + RULD2B[i], RULDColor[i]);
		}
	}
	Reset();
	t = clock();
}

RubiksCube::~RubiksCube() {
	delete[] this->data;
}

void RubiksCube::Reset() {
	this->htm = 0;
	this->qtm = 0;
	this->lastFace = -1;
	memset(this->data + size * size * 0, 0, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 1, 1, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 2, 2, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 3, 3, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 4, 4, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 5, 5, sizeof(unsigned char) * size * size);
}

void RubiksCube::Shuffle(int time) {
	for (size_t i = 0; i < time; i++)
	{
		int type = rand() % 6;
		int column = (rand() % (int)(size /2))+1;

		int angle = (rand() % 3) + 1;
		Rotate(type, column, angle);
	}
}

void RubiksCube::Solve() {
	int step = 0;
	t = clock();
	if (size == 2) {
		std::vector<int> types{ 0,1,2 };
		for (size_t i = 0; i < 500; i++)
		{
			t = clock();
			Shuffle(200);
			int phase = 1;
			//solve to lasted 2 step
			if (!directSearch2x2Tree(1, 9, -1, types, phase)) {
				std::cout << "faild phase 1 QQ" << std::endl;
				break;
			}
			else {
				phase++;
				if(!directSearch2x2Tree(1, 3, -1, types, phase)) {
					std::cout << "faild phase 2 QQ" << std::endl;
					break;
				}
			}
			//solve final 2 step;
			clock_t pass = clock() - t;
			std::cout << "pass time: " << (float)pass / CLOCKS_PER_SEC << " second." << std::endl;
		}
		return;
	}
	//condition.size()/2
	//for (size_t i = 0; i < condition.size() / 2; i++)
	//{
	//	if (!SolveTree(1, 6, -1,i)) {
	//		std::cout <<"step "<< i << " fail" << std::endl;
	//		break;
	//		Shuffle(6);
	//		//magicTreak();
	//		i = -1;
	//		continue;
	//	}
	//	std::cout << "step "<< i <<" pass" << std::endl;
	//}
	//std::cout << "success" << std::endl;
	//step 1,2 solve D layer cross,conor
	//we belive that we can do least 6 move search to complete this set
	//<= condition.size()/2
	for (size_t i = 0; i <= 21; i++)
	{
		if (!SolveTree(1, 6, -1, i)) {
			std::cout << "step " << i << " fail" << std::endl;
			break;
		}
		step = i;
		std::cout << "step " << i << " pass" << std::endl;
	}
	//step 3 solve middle layer coner
	std::cout << "solve middle " << std::endl;
	for (size_t i = step; i <= 29; i++)
	{
		if (!SolveMiddle(1, 3, -1, i)) {
			std::cout << "step " << i << " fail" << std::endl;
			break;
		}
		step = i;
		std::cout << "step " << i << " pass" << std::endl;
	}
	//step 4 B cross face
	std::cout << "solve B cross" << std::endl;
	for (size_t i = step; i <= 33; i++)
	{
		if (!SolveBCross(1, 3, -1, i)) {
			std::cout << "step " << i << " fail" << std::endl;
			break;
		}
		step = i;
		std::cout << "step " << i << " pass" << std::endl;
	}
	//step 5 B Corner face
	if (!SolveBCorner(1, 4, -1, 37)) {
		std::cout << "solve BCorner fail" << std::endl;
	}
	else {
		std::cout << "BCorner pass" << std::endl;
	}
	//step6 B Coner Swape
	if (!SolveBCornerSwap(1, 3, -1, 37 + 8)) {
		std::cout << "solve BCornerSwap fail" << std::endl;
	}
	else {
		std::cout << "solve BCornerSwap pass" << std::endl;
	}
	//step7 B Cross swapeSolveBCrossSwap
	if (!SolveBCrossSwap(1,2, -1, 37 + 8+4)) {
		std::cout << "solve BCrossSwap fail" << std::endl;
	}
	else {
		std::cout << "solve BCrossSwap pass" << std::endl;
	}
	if (isSolved()) {
		std::cout << "solved!!" << std::endl;
	}
}
bool RubiksCube::SolveMiddle(int turn,int maxTurn,int lastFace,int step) {
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t angle = 0; angle < angles.size(); angle++)
		{
			for (size_t j = 0; j < 2; j++)
			{
				Rotate(U, column, angles[angle]);
				if (j == 0) {
					Rotate(U, column, 1);
					Rotate(R, column, 1);
					Rotate(U, column, 3);
					Rotate(R, column, 3);
					Rotate(U, column, 3);
					Rotate(F, column, 3);
					Rotate(U, column, 1);
					Rotate(F, column, 1);
				}
				else if (j == 1) {
					Rotate(U, column, 3);
					Rotate(L, column, 3);
					Rotate(U, column, 1);
					Rotate(L, column, 1);
					Rotate(U, column, 1);
					Rotate(F, column, 1);
					Rotate(U, column, 3);
					Rotate(F, column, 3);
				}
				if (SolveMiddle(turn + 1, maxTurn, -1, step))
					return true;
				else {
					if (j == 0) {
						Rotate(F, column, 3);
						Rotate(U, column, 3);
						Rotate(F, column, 1);
						Rotate(U, column, 1);
						Rotate(R, column, 1);
						Rotate(U, column, 1);
						Rotate(R, column, 3);
						Rotate(U, column, 3);
					}
					else if (j == 1) {
						Rotate(F, column, 1);
						Rotate(U, column, 1);
						Rotate(F, column, 3);
						Rotate(U, column, 3);
						Rotate(L, column, 3);
						Rotate(U, column, 3);
						Rotate(L, column, 1);
						Rotate(U, column, 1);
					}
					Rotate(U, column, 4 - angles[angle]);
				}
				
			}	
		}
	}
	return false;
}
bool RubiksCube::SolveBCross(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t j = 0; j < 2; j++)
		{
			Rotate(F, column, 1);
			Rotate(R, column, 1);
			Rotate(U, column, 1);
			Rotate(R, column, 3);
			Rotate(U, column, 3);
			Rotate(F, column, 3);
			if (SolveMiddle(turn + 1, maxTurn, -1, step))
				return true;
			else {
				Rotate(F, column, 1);
				Rotate(U, column, 1);
				Rotate(R, column, 1);
				Rotate(U, column, 3);
				Rotate(R, column, 3);
				Rotate(F, column, 3);
			}
		}
	}
	return false;
}
bool RubiksCube::SolveBCorner(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		Rotate(R, column, 3);
		Rotate(U, column, 2);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 3);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		if (SolveBCorner(turn + 1, maxTurn, -1, step))
			return true;
		else {
			Rotate(R, column, 3);
			Rotate(U, column, 3);
			Rotate(R, column, 1);
			Rotate(U, column, 3);
			Rotate(R, column, 3);
			Rotate(U, column, 2);
			Rotate(R, column, 1);
		}
	}
	return false;
}
bool RubiksCube::SolveBCornerSwap(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int B = table[(i + 2) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t angle = 0; angle < angles.size(); angle++)
		{
			Rotate(U, column, angles[angle]);
			Rotate(R, column, 3);
			Rotate(F, column, 1);
			Rotate(R, column, 3);
			Rotate(B, column, 2);
			Rotate(R, column, 1);
			Rotate(F, column, 3);
			Rotate(R, column, 3);
			Rotate(B, column, 2);
			Rotate(R, column, 2);
			if (SolveBCornerSwap(turn + 1, maxTurn, -1, step))
				return true;
			else {
				Rotate(R, column, 2);
				Rotate(B, column, 2);
				Rotate(R, column, 1);
				Rotate(F, column, 1);
				Rotate(R, column, 3);
				Rotate(B, column, 2);
				Rotate(R, column, 1);
				Rotate(F, column, 3);
				Rotate(R, column, 1);
				Rotate(U, column, 4 - angles[angle]);
			}
			
			
		}
		
	}
	return false;
}
bool RubiksCube::SolveBCrossSwap(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		Rotate(R, column, 1);
		Rotate(U, column, 3);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		Rotate(U, column, 3);
		Rotate(R, column, 3);
		Rotate(U, column, 3);
		Rotate(R, column, 2);
		if (SolveBCrossSwap(turn + 1, maxTurn, -1, step))
			return true;
		else {
			Rotate(R, column, 2);
			Rotate(U, column, 1);
			Rotate(R, column, 1);
			Rotate(U, column, 1);
			Rotate(R, column, 3);
			Rotate(U, column, 3);
			Rotate(R, column, 3);
			Rotate(U, column, 3);
			Rotate(R, column, 3);
			Rotate(U, column, 1);
			Rotate(R, column, 3);
		}
	}
	return false;
}
bool RubiksCube::SolveTree(int turn, int maxTurn,int lastFace,int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> types{1,2,3,4,5};//never do the F TURN
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(types.begin(), types.end());
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t type = 0; type < types.size(); type++)
	{
		if (lastFace == types[type])
			continue;
		for (size_t column = 1; column <= (int)(size/2); column++)
		{
			for (size_t angle = 0; angle < angles.size(); angle++)
			{
				Rotate(types[type], column, angles[angle]);
				if (SolveTree(turn + 1, maxTurn, types[type], step))
					return true;
				else
					Rotate(types[type], column, 4 - angles[angle]);
			}
		}
	}
	return false;
}

int RubiksCube::rank2x2()
{
	if (size != 2)return 0;
	int rank = 0;
	int a = 0, b = 0;
	for (size_t i = 0; i < 6; i++)
	{
		//
		a = 0; b = 0;
		a += data[size*size*i + 0] == data[size*size*i + 1] ? 1 : 0;
		a += data[size*size*i + 2] == data[size*size*i + 3] ? 1 : 0;
		b += data[size*size*i + 0] == data[size*size*i + 2] ? 1 : 0;
		b += data[size*size*i + 1] == data[size*size*i + 3] ? 1 : 0;
		if (a == b&&a == 2)
			rank += 100;
		rank += a > b ? a : b;
	}
	return rank;
}

void RubiksCube::ConditionPush(std::vector<int> &v, int index, int color)
{
	v.push_back(index);
	v.push_back(color);
}

void RubiksCube::Rotate(OperateType type, int column, int angle) {
	Rotate((int)type, angle, column);
}

// ��x�}����M��g��t�@�ӯx�}
void RotateArray(unsigned char *dst, unsigned char *src, int size, int angle) {
	angle = (angle % 4 + 4) % 4;
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			int index = y * size + x;
			int newIndex = 0;
			if (angle == 0) newIndex = y * size + x;
			if (angle == 1) newIndex = x * size + (size - 1 - y);
			if (angle == 2) newIndex = (size - 1 - y) * size + (size - 1 - x);
			if (angle == 3) newIndex = (size - 1 - x) * size + y;
			dst[newIndex] = src[index];
		}
	}
}

// �����C�ӭ��P�򦳭��X�ӭ�(���ɰw�A�W�k�U��)
static const int relatedFace[6][4] = {
	{ 2, 1, 5, 4 },
	{ 2, 3, 5, 0 },
	{ 3, 1, 0, 4 },
	{ 2, 4, 5, 1 },
	{ 2, 0, 5, 3 },
	{ 0, 1, 3, 4 },
};

// �����C�ӭ��P�򪺭��P�ۤv�۳s����V
static const int relatedFaceDirection[6][4] = {
	{ 2, 3, 0, 1 },
	{ 1, 3, 1, 1 },
	{ 0, 0, 0, 0 },
	{ 0, 3, 2, 1 },
	{ 3, 3, 3, 1 },
	{ 2, 2, 2, 2 },
};

void RubiksCube::Rotate(int type, int column, int angle) {
	htm += lastFace != type;
	lastFace = type;
	qtm += (angle == 2) + 1;
	// �����ӭ�
	unsigned char *buffer = new unsigned char[size * size];
	RotateArray(buffer, data + size * size * type, size, angle);
	memcpy(data + (size * size * type), buffer, sizeof(unsigned char) * size * size);
	delete[] buffer;
	// ����P���۾F����L�����C
	unsigned char *buffers[4];
	// �N��ӭ������s��buffers��
	for (int i = 0; i < 4; i++) {
		int dst = (i + angle) % 4;
		buffers[dst] = new unsigned char[size * size];
		RotateArray(buffers[dst], data + size * size * relatedFace[type][i], size, relatedFaceDirection[type][dst] - relatedFaceDirection[type][i]);
	}
	// �̾�column�N�A�q����Ʀs�^data��
	for (int i = 0; i < 4; i++) {
		if (relatedFaceDirection[type][i] == 0) {
			memcpy(data + size * size * relatedFace[type][i], buffers[i], sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 1) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y + (size - column), buffers[i] + size * y + (size - column), sizeof(unsigned char) * column);
			}
		} else if (relatedFaceDirection[type][i] == 2) {
			memcpy(data + size * size * relatedFace[type][i] + size * (size - column), buffers[i] + size * (size - column), sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 3) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y, buffers[i] + size * y, sizeof(unsigned char) * column);
			}
		}
		delete[] buffers[i];
	}
}

void RubiksCube::Redo() {
	// TODO
}

void RubiksCube::Undo() {
	// TODO
}

bool RubiksCube::isSolved()
{
	int tables[6];
	for (size_t i = 0; i < 6; i++)
	{
		tables[i] = data[size*size * i];
	}
	for (size_t i = 0; i < size; i++)
	{
		for (size_t j = 0; j < size; j++)
		{
			if (data[size*size * 0 + size*i + j] != tables[0])return false;
			if (data[size*size * 1 + size*i + j] != tables[1])return false;
			if (data[size*size * 2 + size*i + j] != tables[2])return false;
			if (data[size*size * 3 + size*i + j] != tables[3])return false;
			if (data[size*size * 4 + size*i + j] != tables[4])return false;
			if (data[size*size * 5 + size*i + j] != tables[5])return false;
		}
	}
	return true;
}

bool RubiksCube::isSolvedPart(int step)
{
	if (size == 2) {
		bool flag = true;
		for (size_t i = 0; i < step; i++)
		{
			if (data[condition[i * 2]] != condition[i * 2 + 1]) {
				flag = false;
				break;
			}
		}
		return flag;
	}
	if (size == 3) {
		bool flag = true;
		for (size_t i = 0; i < step; i++)
		{
			if (data[condition[i * 2]] != condition[i * 2 + 1]) {
				flag = false;
				break;
			}
		}
		return flag;
	}
	return false;
}

bool RubiksCube::checkMe()
{
	int t[100];
	int a[100];
	int c[100];
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		t[i] = rand() % 6;
		a[i] = (rand() % 3) + 1;
		c[i] = (rand() % (size - 1)) + 1;
		//	std::cout << cube;
	}
	for (size_t i = 0; i < 100; i++)
	{
		Rotate(t[i], c[i], a[i]);
	}
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		int index = 99 - i;
		Rotate(t[index], c[index], 4 - a[index]);
	}
	std::cout << *this;
	return false;
}

bool RubiksCube::directSearch2x2Tree(int turn, int maxTurn, int lastFace, std::vector<int> types,int phase)
{
	if (phase == 1) {
		if (rank2x2() == 212)//in that condition we can solve in 4 step;
			return true;
	}else if(phase == 2)
		if (isSolved())//in that condition we can solve in 4 step;
			return true;
	if (turn > maxTurn)
		return false;
	//clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	//if ((float)pass / CLOCKS_PER_SEC > 120) {
	//	return false;
	//}
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(types.begin(), types.end());
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t type = 0; type < types.size(); type++)
	{
		if (lastFace == types[type])
			continue;
		for (size_t column = 1; column <= (int)(size / 2); column++)
		{
			for (size_t angle = 0; angle < angles.size(); angle++)
			{
				Rotate(types[type], column, angles[angle]);
				if (directSearch2x2Tree(turn + 1, maxTurn, types[type], types,phase))
					return true;
				else
					Rotate(types[type], column, 4 - angles[angle]);
			}
		}
	}
	return false;
}

std::ostream& operator<<(std::ostream& outputStream, const RubiksCube& cube) {
	static const char color[] = { 'G', 'R', 'W', 'B', 'O', 'Y' };
	std::string padding(cube.size * 2, ' ');
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 2 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 4 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 0 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 1 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 3 + cube.size * i + j]] << ' ';
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 5 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	outputStream << "htm: " << cube.htm << ", qtm: " << cube.qtm << std::endl;
	return outputStream;
}
