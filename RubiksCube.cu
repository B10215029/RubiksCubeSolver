#include "hip/hip_runtime.h"
#include "RubiksCube.h"
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <GL/glew.h>
#include <cuda_gl_interop.h>

#define THREAD_NUM 1024
#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
	}\
}
__device__ __host__ int CeilDiv(int a, int b) { return (a - 1) / b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }
void CHECK_CUDA(hipError_t err) {
	if (err != hipSuccess) {
		std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
	}
}

RubiksCube::RubiksCube(int size) {
	this->size = size;
	this->data = new unsigned char[size * size * 6];
	useHost = size < 100;
	cudaDataIndex = 0;
	cudaImageArray = NULL;
	for (int i = 0; i < CUDA_DATA_LEN; i++) {
		hipMalloc(cudaData + i, sizeof(unsigned char) * size * size * 6);
	}
	if (size == 2) {
		//first face
		condition.push_back(0);
		condition.push_back(0);
		condition.push_back(1);
		condition.push_back(0);
		condition.push_back(2);
		condition.push_back(0);
		condition.push_back(3);
		condition.push_back(0);
		//up edge
		condition.push_back(10);
		condition.push_back(2);
		condition.push_back(11);
		condition.push_back(2);
		//left edge
		condition.push_back(17);
		condition.push_back(4);
		condition.push_back(19);
		condition.push_back(4);
		//down edge
		condition.push_back(20);
		condition.push_back(5);
		condition.push_back(21);
		condition.push_back(5);
		//right edge
		condition.push_back(4);
		condition.push_back(1);
		condition.push_back(6);
		condition.push_back(1);
		
		//up back edge
		condition.push_back(8);
		condition.push_back(2);
		condition.push_back(9);
		condition.push_back(2);
		//left back edge
		condition.push_back(16);
		condition.push_back(4);
		condition.push_back(18);
		condition.push_back(4);
		//down back edge
		condition.push_back(22);
		condition.push_back(5);
		condition.push_back(23);
		condition.push_back(5);
		//right back edge
		condition.push_back(5);
		condition.push_back(1);
		condition.push_back(7);
		condition.push_back(1);
		//back face
		condition.push_back(12);
		condition.push_back(3);
		condition.push_back(13);
		condition.push_back(3);
		condition.push_back(14);
		condition.push_back(3);
		condition.push_back(15);
		condition.push_back(3);
	}
	if (false) {
		int RULDIndex[4] = { 13,22,40,49 };
		int RULDColor[4] = { 1,2,4,5 };
		int crossIndex[4] = { -3,1,3,-1 };
		int cornerIndex[4] = { -4,-2,4,2 };
		int edgeLeft[3] = { -4,-1,2 };
		int edgeRight[3] = { -2,1,4 };
		int edgeUp[3] = { -4,-3,-2 };
		int edgeDown[3] = { 2,3,4 };
		int UDL [3] = { -3,0,3 };
		int RLL [3] = { -1,0,1 };
		int faceIndex = 4;
		int faceColor = 0;
		//F center
		condition.push_back(faceIndex);
		condition.push_back(0);
		//F cross
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
		}
		//F corner
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
		}
		//R edge
		faceColor = 1;
		faceIndex = 13;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeLeft[i], faceColor);
		}
		//U edge
		faceColor = 2;
		faceIndex = 22;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeDown[i], faceColor);
		}
		//L edge
		faceColor = 4;
		faceIndex = 40;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeRight[i], faceColor);
		}
		//D edge
		faceColor = 5;
		faceIndex = 49;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + edgeUp[i], faceColor);
		}
		//ring center
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, RULDIndex[i], RULDColor[i]);
		}
		//R middle
		faceColor = 1;
		faceIndex = 13;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + UDL[i], faceColor);
		}
		//U middle
		faceColor = 2;
		faceIndex = 22;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + RLL[i], faceColor);
		}
		//L middle
		faceColor = 4;
		faceIndex = 40;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + UDL[i], faceColor);
		}
		//D middle
		faceColor = 5;
		faceIndex = 49;
		for (size_t i = 0; i < 3; i++)
		{
			ConditionPush(condition, faceIndex + RLL[i], faceColor);
		}
	}
	if (size == 3) {
		int RULDIndex[4] = { 13,22,40,49 };
		int RULDColor[4] = { 1,2,4,5 };
		int RULD2F[4] = { -1,3,1,-3 };
		int RULD2B[4] = { 1,-3,-1,3 };
		int RULD2FRC[4] = { 2,4,-2,-4 };
		int RULD2FLC[4] = { -4,2,4,-2 };
		int RULD2MLE[4] = {-3,-1,3,1};
		int RULD2MRE[4] = { 3,1,-3,-1 };
		int RULD2BRC[4] = {4,-2,-4,2};
		int RULD2BLC[4] = {-2,-4,2,4};

		int crossIndex[4] = { 1,-3,-1,3};
		int cornerIndex[4] = { -2,-4,2,4 };
		int edgeLeft[3] = { -4,-1,2 };
		int edgeRight[3] = { -2,1,4 };
		int edgeUp[3] = { -4,-3,-2 };
		int edgeDown[3] = { 2,3,4 };
		int UDL[3] = { -3,0,3 };
		int RLL[3] = { -1,0,1 };
		int faceIndex = 4;
		int faceColor = 0;
		//F center
		condition.push_back(faceIndex);
		condition.push_back(0);
		//F cross
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
			ConditionPush(condition, RULDIndex[i]+ RULD2F[i], RULDColor[i]);
		}
		//F conor
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
			ConditionPush(condition, RULDIndex[i] + RULD2FLC[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2FRC[plusIndex], RULDColor[plusIndex]);
		}
		//middle conor
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, RULDIndex[i] + RULD2MLE[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2MRE[plusIndex], RULDColor[plusIndex]);
		}
		//B cross Face
		faceIndex = 31;
		faceColor = 3;
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + crossIndex[i], faceColor);
		}
		//B corner Face
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, faceIndex + cornerIndex[i], faceColor);
		}
		//RULD B Conr Face
		for (size_t i = 0; i < 4; i++)
		{
			int plusIndex = (i + 1) % 4;
			ConditionPush(condition, RULDIndex[i] + RULD2BLC[i], RULDColor[i]);
			ConditionPush(condition, RULDIndex[plusIndex] + RULD2BRC[plusIndex], RULDColor[plusIndex]);
		}
		//RULD B Cross Face
		for (size_t i = 0; i < 4; i++)
		{
			ConditionPush(condition, RULDIndex[i] + RULD2B[i], RULDColor[i]);
		}
	}
	Reset();
	t = clock();
}

RubiksCube::~RubiksCube() {
	delete[] this->data;
	for (int i = 0; i < CUDA_DATA_LEN; i++) {
		hipFree(cudaData[i]);
	}
}

__global__ void CudaReset(unsigned char* data, int size) {
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned char type = index / (size * size);
	if (type < 6) {
		data[index] = type;
	}
}

void RubiksCube::MapTexture(unsigned char textureID) {
	hipGraphicsResource *cudaImageResource;
	CHECK_CUDA(hipGraphicsGLRegisterImage(&cudaImageResource, textureID, GL_TEXTURE_CUBE_MAP, hipGraphicsRegisterFlagsSurfaceLoadStore));
	CHECK_CUDA(hipGraphicsMapResources(1, &cudaImageResource, 0));
	CHECK_CUDA(hipGraphicsSubResourceGetMappedArray(&cudaImageArray, cudaImageResource, 0, 0));
}

void RubiksCube::Reset() {
	this->htm = 0;
	this->qtm = 0;
	this->lastFace = -1;
	if (useHost) {
		memset(this->data + size * size * 0, 0, sizeof(unsigned char) * size * size);
		memset(this->data + size * size * 1, 1, sizeof(unsigned char) * size * size);
		memset(this->data + size * size * 2, 2, sizeof(unsigned char) * size * size);
		memset(this->data + size * size * 3, 3, sizeof(unsigned char) * size * size);
		memset(this->data + size * size * 4, 4, sizeof(unsigned char) * size * size);
		memset(this->data + size * size * 5, 5, sizeof(unsigned char) * size * size);
	} else {
		CudaReset << <dim3(CeilDiv(size * size * 6, THREAD_NUM)), dim3(THREAD_NUM) >> >(cudaData[0], size);
		//for (int i = 1; i < CUDA_DATA_LEN; i++) {
		//	hipMemcpy(cudaData[i], cudaData[0], sizeof(unsigned char) * size * size * 6, hipMemcpyDeviceToDevice);
		//}
		//hipMemcpy(this->data, cudaData[0], sizeof(unsigned char) * size * size * 6, hipMemcpyDeviceToHost);
	}
	turnNumber = 0;
	turnStacks.clear();
}

void RubiksCube::Shuffle(int time) {
	for (size_t i = 0; i < time; i++)
	{
		int type = rand() % 6;
		int column = (rand() % (int)(size /2))+1;

		int angle = (rand() % 3) + 1;
		Rotate(type, column, angle);
	}
}

void RubiksCube::Solve() {
	turnStacks.clear();
	turnNumber = 0;
	int step = 0;
	t = clock();
	if (size == 2) {
		std::vector<int> types{ 0,1,2 };
		int phase = 1;
		//solve to lasted 2 step
		if (!directSearch2x2Tree(1, 9, -1, types, phase)) {
			std::cout << "faild phase 1 QQ" << std::endl;
		}
		else {
			phase++;
			if (!directSearch2x2Tree(1, 3, -1, types, phase)) {
				std::cout << "faild phase 2 QQ" << std::endl;
			}
		}
		clock_t pass = clock() - t;
		std::cout << "pass time: " << (float)pass / CLOCKS_PER_SEC << " second." << std::endl;
		std::cout << "Stack turn: " << turnNumber << std::endl;
		return;
	}
	else if (size == 3) {
		//simple search in depth 6
		std::vector<int> limit_t{5,4,3,2,1,0 };
		SolveTree(1, 5, -1, condition.size() / 2, limit_t);
		limit_t.pop_back();
		for (size_t i = 0; i <= 21; i++)
		{
			if (!SolveTree(1, 6, -1, i, limit_t)) {
				std::cout << "step " << i << " fail" << std::endl;
				break;
			}
			step = i;
			std::cout << "step " << i << " pass" << std::endl;
		}
		//step 3 solve middle layer coner
		std::cout << "solve middle " << std::endl;
		for (size_t i = step; i <= 29; i++)
		{
			if (!SolveMiddle(1, 3, -1, i)) {
				std::cout << "step " << i << " fail" << std::endl;
				break;
			}
			step = i;
			std::cout << "step " << i << " pass" << std::endl;
		}
		//step 4 B cross face
		std::cout << "solve B cross" << std::endl;
		for (size_t i = step; i <= 33; i++)
		{
			if (!SolveBCross(1, 3, -1, i)) {
				std::cout << "step " << i << " fail" << std::endl;
				break;
			}
			step = i;
			std::cout << "step " << i << " pass" << std::endl;
		}
		//step 5 B Corner face
		if (!SolveBCorner(1, 4, -1, 37)) {
			std::cout << "solve BCorner fail" << std::endl;
		}
		else {
			std::cout << "BCorner pass" << std::endl;
		}
		//step6 B Coner Swape
		if (!SolveBCornerSwap(1, 3, -1, 37 + 8)) {
			std::cout << "solve BCornerSwap fail" << std::endl;
		}
		else {
			std::cout << "solve BCornerSwap pass" << std::endl;
		}
		//step7 B Cross swapeSolveBCrossSwap
		if (!SolveBCrossSwap(1, 2, -1, 37 + 8 + 4)) {
			std::cout << "solve BCrossSwap fail" << std::endl;
		}
		else {
			std::cout << "solve BCrossSwap pass" << std::endl;
		}
		if (isSolved()) {
			std::cout << "solved!!" << std::endl;
		}
		clock_t pass = clock() - t;
		std::cout << "pass time: " << (float)pass / CLOCKS_PER_SEC << " second." << std::endl;
		std::cout << "Stack turn: " << turnNumber << std::endl;
	}
	
}
bool RubiksCube::SolveMiddle(int turn,int maxTurn,int lastFace,int step) {
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t angle = 0; angle < angles.size(); angle++)
		{
			for (size_t j = 0; j < 2; j++)
			{
				Rotate(U, column, angles[angle]);
				if (j == 0) {
					Rotate(U, column, 1);
					Rotate(R, column, 1);
					Rotate(U, column, 3);
					Rotate(R, column, 3);
					Rotate(U, column, 3);
					Rotate(F, column, 3);
					Rotate(U, column, 1);
					Rotate(F, column, 1);
				}
				else if (j == 1) {
					Rotate(U, column, 3);
					Rotate(L, column, 3);
					Rotate(U, column, 1);
					Rotate(L, column, 1);
					Rotate(U, column, 1);
					Rotate(F, column, 1);
					Rotate(U, column, 3);
					Rotate(F, column, 3);
				}
				if (SolveMiddle(turn + 1, maxTurn, -1, step))
					return true;
				else {
					if (j == 0) {
						for (size_t turnTimes = 0; turnTimes < 8; turnTimes++)
						{
							Undo();
						}
						//Rotate(F, column, 3);
						//Rotate(U, column, 3);
						//Rotate(F, column, 1);
						//Rotate(U, column, 1);
						//Rotate(R, column, 1);
						//Rotate(U, column, 1);
						//Rotate(R, column, 3);
						//Rotate(U, column, 3);
					}
					else if (j == 1) {
						for (size_t turnTimes = 0; turnTimes < 8; turnTimes++)
						{
							Undo();
						}
						//Rotate(F, column, 1);
						//Rotate(U, column, 1);
						//Rotate(F, column, 3);
						//Rotate(U, column, 3);
						//Rotate(L, column, 3);
						//Rotate(U, column, 3);
						//Rotate(L, column, 1);
						//Rotate(U, column, 1);
					}
					Undo();
					//Rotate(U, column, 4 - angles[angle]);
				}
				
			}	
		}
	}
	return false;
}
bool RubiksCube::SolveBCross(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t j = 0; j < 2; j++)
		{
			Rotate(F, column, 1);
			Rotate(R, column, 1);
			Rotate(U, column, 1);
			Rotate(R, column, 3);
			Rotate(U, column, 3);
			Rotate(F, column, 3);
			if (SolveMiddle(turn + 1, maxTurn, -1, step))
				return true;
			else {
				for (size_t turnTimes = 0; turnTimes < 6; turnTimes++)
				{
					Undo();
				}
				//Rotate(F, column, 1);
				//Rotate(U, column, 1);
				//Rotate(R, column, 1);
				//Rotate(U, column, 3);
				//Rotate(R, column, 3);
				//Rotate(F, column, 3);
			}
		}
	}
	return false;
}
bool RubiksCube::SolveBCorner(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		Rotate(R, column, 3);
		Rotate(U, column, 2);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 3);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		if (SolveBCorner(turn + 1, maxTurn, -1, step))
			return true;
		else {
			for (size_t turnTimes = 0; turnTimes < 7; turnTimes++)
			{
				Undo();
			}
			//Rotate(R, column, 3);
			//Rotate(U, column, 3);
			//Rotate(R, column, 1);
			//Rotate(U, column, 3);
			//Rotate(R, column, 3);
			//Rotate(U, column, 2);
			//Rotate(R, column, 1);
		}
	}
	return false;
}
bool RubiksCube::SolveBCornerSwap(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int B = table[(i + 2) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		for (size_t angle = 0; angle < angles.size(); angle++)
		{
			Rotate(U, column, angles[angle]);
			Rotate(R, column, 3);
			Rotate(F, column, 1);
			Rotate(R, column, 3);
			Rotate(B, column, 2);
			Rotate(R, column, 1);
			Rotate(F, column, 3);
			Rotate(R, column, 3);
			Rotate(B, column, 2);
			Rotate(R, column, 2);
			if (SolveBCornerSwap(turn + 1, maxTurn, -1, step))
				return true;
			else {
				for (size_t turnTimes = 0; turnTimes < 10; turnTimes++)
				{
					Undo();
				}
				//Rotate(R, column, 2);
				//Rotate(B, column, 2);
				//Rotate(R, column, 1);
				//Rotate(F, column, 1);
				//Rotate(R, column, 3);
				//Rotate(B, column, 2);
				//Rotate(R, column, 1);
				//Rotate(F, column, 3);
				//Rotate(R, column, 1);
				//Rotate(U, column, 4 - angles[angle]);
			}
			
			
		}
		
	}
	return false;
}
bool RubiksCube::SolveBCrossSwap(int turn, int maxTurn, int lastFace, int step)
{
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> table{ 4,2,1,5 };
	for (size_t i = 0; i < table.size(); i++)
	{
		int R = table[(i + 1) % table.size()];
		int F = table[i];
		int L = table[(i - 1 + table.size()) % table.size()];
		int U = 3;
		int D = 0;
		int column = 1;
		Rotate(R, column, 1);
		Rotate(U, column, 3);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		Rotate(U, column, 1);
		Rotate(R, column, 1);
		Rotate(U, column, 3);
		Rotate(R, column, 3);
		Rotate(U, column, 3);
		Rotate(R, column, 2);
		if (SolveBCrossSwap(turn + 1, maxTurn, -1, step))
			return true;
		else {
			for (size_t turnTimes = 0; turnTimes < 11; turnTimes++)
			{
				Undo();
			}
			//Rotate(R, column, 2);
			//Rotate(U, column, 1);
			//Rotate(R, column, 1);
			//Rotate(U, column, 1);
			//Rotate(R, column, 3);
			//Rotate(U, column, 3);
			//Rotate(R, column, 3);
			//Rotate(U, column, 3);
			//Rotate(R, column, 3);
			//Rotate(U, column, 1);
			//Rotate(R, column, 3);
		}
	}
	return false;
}
bool RubiksCube::SolveTree(int turn, int maxTurn,int lastFace,int step, std::vector<int> limit_t)
{
	if (isSolved())
		return true;
	if (isSolvedPart(step))
		return true;
	if (turn > maxTurn)
		return false;
	clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	if ((float)pass / CLOCKS_PER_SEC > 120) {
		return false;
	}
	std::vector<int> types = limit_t;
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(types.begin(), types.end());
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t type = 0; type < types.size(); type++)
	{
		if (lastFace == types[type])
			continue;
		for (size_t column = 1; column <= (int)(size/2); column++)
		{
			for (size_t angle = 0; angle < angles.size(); angle++)
			{
				Rotate(types[type], column, angles[angle]);
				if (SolveTree(turn + 1, maxTurn, types[type], step, limit_t))
					return true;
				else {
					// Rotate(types[type], column, 4 - angles[angle]);
					Undo();
				}
					
			}
		}
	}
	return false;
}

int RubiksCube::rank2x2()
{
	if (size != 2)return 0;
	int rank = 0;
	int a = 0, b = 0;
	for (size_t i = 0; i < 6; i++)
	{
		//
		a = 0; b = 0;
		a += data[size*size*i + 0] == data[size*size*i + 1] ? 1 : 0;
		a += data[size*size*i + 2] == data[size*size*i + 3] ? 1 : 0;
		b += data[size*size*i + 0] == data[size*size*i + 2] ? 1 : 0;
		b += data[size*size*i + 1] == data[size*size*i + 3] ? 1 : 0;
		if (a == b&&a == 2)
			rank += 100;
		rank += a > b ? a : b;
	}
	return rank;
}

void RubiksCube::ConditionPush(std::vector<int> &v, int index, int color)
{
	v.push_back(index);
	v.push_back(color);
}

void RubiksCube::Rotate(OperateType type, int column, int angle) {
	Rotate((int)type, angle, column);
	
}

__global__ void CudaRotate(unsigned char *dst, unsigned char *src, int size, int type, int column, int angle) {
	// �����C�ӭ��P�򦳭��X�ӭ�(���ɰw�A�W�k�U���e��)
	static const int relatedFace[6][6] = {
		{ 2, 1, 5, 4, 0, 3 },
		{ 2, 3, 5, 0, 1, 4 },
		{ 3, 1, 0, 4, 2, 5 },
		{ 2, 4, 5, 1, 3, 0 },
		{ 2, 0, 5, 3, 4, 1 },
		{ 0, 1, 3, 4, 5, 2 },
	};
	// �����C�ӭ��P�򪺭��P�ۤv�۳s����V
	static const int relatedFaceDirection[6][4] = {
		{ 2, 3, 0, 1 },
		{ 1, 3, 1, 1 },
		{ 0, 0, 0, 0 },
		{ 0, 3, 2, 1 },
		{ 3, 3, 3, 1 },
		{ 2, 2, 2, 2 },
	};
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned char indexType = index / (size * size);
	if (indexType < 6) {
		const int typeIndex = index % (size * size);
		const int x = typeIndex % size;
		const int y = typeIndex / size;
		unsigned char srcFace, dstFace;
		srcFace = dstFace = relatedFace[type][indexType];
		bool skip = false;

		if (indexType < 4) {
			dstFace = relatedFace[type][(indexType + angle) % 4];
			angle = relatedFaceDirection[type][(indexType + angle) % 4] - relatedFaceDirection[type][indexType];
			if (relatedFaceDirection[type][indexType] == 0) {
				skip = y >= column;
			}
			else if (relatedFaceDirection[type][indexType] == 1) {
				skip = x < size - column;
			}
			else if (relatedFaceDirection[type][indexType] == 2) {
				skip = y < size - column;
			}
			else if (relatedFaceDirection[type][indexType] == 3) {
				skip = x >= column;
			}
		}
		//else if (indexType == 4) {
		//	angle = angle;
		//}
		else if (indexType == 5) {
			angle = 4 - angle;
			skip = size != column;
		}

		int newIndex = index = size * size * srcFace + typeIndex;
		if (!skip) {
			newIndex = size * size * dstFace;
			angle = (angle % 4 + 4) % 4;
			if (angle == 0) newIndex += typeIndex; // y * size + x;
			if (angle == 1) newIndex += x * size + (size - 1 - y);
			if (angle == 2) newIndex += (size - 1 - y) * size + (size - 1 - x);
			if (angle == 3) newIndex += (size - 1 - x) * size + y;
		}
		dst[newIndex] = src[index];
	}
}

// ��x�}����M��g��t�@�ӯx�}
void RotateArray(unsigned char *dst, unsigned char *src, int size, int angle) {
	angle = (angle % 4 + 4) % 4;
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			int index = y * size + x;
			int newIndex = 0;
			if (angle == 0) newIndex = y * size + x;
			if (angle == 1) newIndex = x * size + (size - 1 - y);
			if (angle == 2) newIndex = (size - 1 - y) * size + (size - 1 - x);
			if (angle == 3) newIndex = (size - 1 - x) * size + y;
			dst[newIndex] = src[index];
		}
	}
}

// �����C�ӭ��P�򦳭��X�ӭ�(���ɰw�A�W�k�U��)
static const int relatedFace[6][4] = {
	{ 2, 1, 5, 4 },
	{ 2, 3, 5, 0 },
	{ 3, 1, 0, 4 },
	{ 2, 4, 5, 1 },
	{ 2, 0, 5, 3 },
	{ 0, 1, 3, 4 },
};

// �����C�ӭ��P�򪺭��P�ۤv�۳s����V
static const int relatedFaceDirection[6][4] = {
	{ 2, 3, 0, 1 },
	{ 1, 3, 1, 1 },
	{ 0, 0, 0, 0 },
	{ 0, 3, 2, 1 },
	{ 3, 3, 3, 1 },
	{ 2, 2, 2, 2 },
};

void RubiksCube::HostRotate(int type, int column, int angle) {
	// �����ӭ�
	unsigned char *buffer = new unsigned char[size * size];
	RotateArray(buffer, data + size * size * type, size, angle);
	memcpy(data + (size * size * type), buffer, sizeof(unsigned char) * size * size);
	delete[] buffer;
	// ����P���۾F����L�����C
	unsigned char *buffers[4];
	// �N��ӭ������s��buffers��
	for (int i = 0; i < 4; i++) {
		int dst = (i + angle) % 4;
		buffers[dst] = new unsigned char[size * size];
		RotateArray(buffers[dst], data + size * size * relatedFace[type][i], size, relatedFaceDirection[type][dst] - relatedFaceDirection[type][i]);
	}
	// �̾�column�N�A�q����Ʀs�^data��
	for (int i = 0; i < 4; i++) {
		if (relatedFaceDirection[type][i] == 0) {
			memcpy(data + size * size * relatedFace[type][i], buffers[i], sizeof(unsigned char) * size * column);
		}
		else if (relatedFaceDirection[type][i] == 1) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y + (size - column), buffers[i] + size * y + (size - column), sizeof(unsigned char) * column);
			}
		}
		else if (relatedFaceDirection[type][i] == 2) {
			memcpy(data + size * size * relatedFace[type][i] + size * (size - column), buffers[i] + size * (size - column), sizeof(unsigned char) * size * column);
		}
		else if (relatedFaceDirection[type][i] == 3) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y, buffers[i] + size * y, sizeof(unsigned char) * column);
			}
		}
		delete[] buffers[i];
	}
}

void RubiksCube::Rotate(int type, int column, int angle, bool unredo) {
	htm += lastFace != type;
	lastFace = type;
	qtm += (angle == 2) + 1;
	if (useHost) {
		HostRotate(type, column, angle);
	} else {
		unsigned char* src = GetCudaData();
		SwitchCudaData();
		unsigned char* dst = GetCudaData();
		CudaRotate << <dim3(CeilDiv(size * size * 6, THREAD_NUM)), dim3(THREAD_NUM) >> >(dst, src, size, type, column, angle);
	}
	if (!unredo) {
		
		turnStacks.resize(turnNumber);
		turnNumber++;
		turnStacks.push_back(turn{ (int)type,angle,column });
	}
	
}

void RubiksCube::Redo() {
	// TODO
	if (turnNumber >= turnStacks.size())
		return;
	Rotate(turnStacks[turnNumber].t, turnStacks[turnNumber].c, turnStacks[turnNumber].a,true);
	//std::cout << turnStacks[turnNumber].t << turnStacks[turnNumber].c << turnStacks[turnNumber].a;
	turnNumber++;
}

void RubiksCube::Undo() {
	// TODO
	if (turnNumber <= 0)
		return;
	turnNumber--;
	Rotate(turnStacks[turnNumber].t, turnStacks[turnNumber].c, 4-turnStacks[turnNumber].a,true);
	//std::cout << turnStacks[turnNumber].t << turnStacks[turnNumber].c << turnStacks[turnNumber].a;
}

unsigned char* RubiksCube::SynchronizeData() const {
	if (useHost) {
		hipMemcpy(GetCudaData(), this->data, sizeof(unsigned char) * size * size * 6, hipMemcpyHostToDevice);
	} else {
		hipMemcpy(this->data, GetCudaData(), sizeof(unsigned char) * size * size * 6, hipMemcpyDeviceToHost);
	}
	if (cudaImageArray) {
		hipMemcpyToArray(cudaImageArray, 0, 0, GetCudaData(), sizeof(unsigned char) * size * size * 6, hipMemcpyDeviceToDevice);
	}
	return data;
}

bool RubiksCube::isSolved()
{
	int tables[6];
	for (size_t i = 0; i < 6; i++)
	{
		tables[i] = data[size*size * i];
	}
	for (size_t i = 0; i < size; i++)
	{
		for (size_t j = 0; j < size; j++)
		{
			if (data[size*size * 0 + size*i + j] != tables[0])return false;
			if (data[size*size * 1 + size*i + j] != tables[1])return false;
			if (data[size*size * 2 + size*i + j] != tables[2])return false;
			if (data[size*size * 3 + size*i + j] != tables[3])return false;
			if (data[size*size * 4 + size*i + j] != tables[4])return false;
			if (data[size*size * 5 + size*i + j] != tables[5])return false;
		}
	}
	return true;
}

bool RubiksCube::isSolvedPart(int step)
{
	if (size == 2) {
		bool flag = true;
		for (size_t i = 0; i < step; i++)
		{
			if (data[condition[i * 2]] != condition[i * 2 + 1]) {
				flag = false;
				break;
			}
		}
		return flag;
	}
	if (size == 3) {
		bool flag = true;
		for (size_t i = 0; i < step; i++)
		{
			if (data[condition[i * 2]] != condition[i * 2 + 1]) {
				flag = false;
				break;
			}
		}
		return flag;
	}
	return false;
}

bool RubiksCube::checkMe()
{
	int t[100];
	int a[100];
	int c[100];
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		t[i] = rand() % 6;
		a[i] = (rand() % 3) + 1;
		c[i] = (rand() % (size - 1)) + 1;
		//	std::cout << cube;
	}
	for (size_t i = 0; i < 100; i++)
	{
		Rotate(t[i], c[i], a[i]);
	}
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		int index = 99 - i;
		Rotate(t[index], c[index], 4 - a[index]);
	}
	std::cout << *this;
	return false;
}

bool RubiksCube::directSearch2x2Tree(int turn, int maxTurn, int lastFace, std::vector<int> types,int phase)
{
	if (phase == 1) {
		if (rank2x2() == 212)//in that condition we can solve in 4 step;
			return true;
	}else if(phase == 2)
		if (isSolved())//in that condition we can solve in 4 step;
			return true;
	if (turn > maxTurn)
		return false;
	//clock_t pass = clock() - t;
	//printf("It took me %d clicks (%f seconds).\n", t, ((float)t) / CLOCKS_PER_SEC);
	//if ((float)pass / CLOCKS_PER_SEC > 120) {
	//	return false;
	//}
	std::vector<int> angles{ 1,2,3 };
	std::random_shuffle(types.begin(), types.end());
	std::random_shuffle(angles.begin(), angles.end());
	for (size_t type = 0; type < types.size(); type++)
	{
		if (lastFace == types[type])
			continue;
		for (size_t column = 1; column <= (int)(size / 2); column++)
		{
			for (size_t angle = 0; angle < angles.size(); angle++)
			{
				Rotate(types[type], column, angles[angle]);
				if (directSearch2x2Tree(turn + 1, maxTurn, types[type], types, phase))
					return true;
				else {
					Undo();
					//Rotate(types[type], column, 4 - angles[angle]);
				}
					
			}
		}
	}
	return false;
}

std::ostream& operator<<(std::ostream& outputStream, const RubiksCube& cube) {
	static const char color[] = { 'G', 'R', 'W', 'B', 'O', 'Y' };
	std::string padding(cube.size * 2, ' ');
	cube.SynchronizeData();
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 2 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 4 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 0 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 1 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 3 + cube.size * i + j]] << ' ';
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 5 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	outputStream << "htm: " << cube.htm << ", qtm: " << cube.qtm << std::endl;
	return outputStream;
}
