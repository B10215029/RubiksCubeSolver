#include "RubiksCube.h"
#include <string>
#include <iostream>

RubiksCube::RubiksCube(int size) {
	this->size = size;
	this->data = new unsigned char[size * size * 6];
	Reset();
}

RubiksCube::~RubiksCube() {
	delete[] this->data;
}

void RubiksCube::Reset() {
	this->htm = 0;
	this->qtm = 0;
	this->lastFace = -1;
	memset(this->data + size * size * 0, 0, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 1, 1, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 2, 2, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 3, 3, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 4, 4, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 5, 5, sizeof(unsigned char) * size * size);
}

void RubiksCube::Shuffle() {
	// TODO
}

void RubiksCube::Solve() {
	// TODO
}

void RubiksCube::Rotate(OperateType type, int column, int angle) {
	Rotate((int)type, angle, column);
}

// ��x�}����M��g��t�@�ӯx�}
void RotateArray(unsigned char *dst, unsigned char *src, int size, int angle) {
	angle = (angle % 4 + 4) % 4;
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			int index = y * size + x;
			int newIndex = 0;
			if (angle == 0) newIndex = y * size + x;
			if (angle == 1) newIndex = x * size + (size - 1 - y);
			if (angle == 2) newIndex = (size - 1 - y) * size + (size - 1 - x);
			if (angle == 3) newIndex = (size - 1 - x) * size + y;
			dst[newIndex] = src[index];
		}
	}
}

// �����C�ӭ��P�򦳭��X�ӭ�(���ɰw�A�W�k�U��)
static const int relatedFace[6][4] = {
	{ 2, 1, 3, 4 },
	{ 2, 5, 3, 0 },
	{ 5, 1, 0, 4 },
	{ 0, 1, 5, 4 },
	{ 2, 0, 3, 5 },
	{ 2, 4, 3, 1 },
};

// �����C�ӭ��P�򪺭��P�ۤv�۳s����V
static const int relatedFaceDirection[6][4] = {
	{ 2, 3, 0, 1 },
	{ 1, 3, 1, 1 },
	{ 0, 0, 0, 0 },
	{ 2, 2, 2, 2 },
	{ 3, 3, 3, 1 },
	{ 0, 3, 2, 1 },
};

void RubiksCube::Rotate(int type, int column, int angle) {
	htm += lastFace != type;
	lastFace = type;
	qtm += (angle == 2) + 1;
	// �����ӭ�
	unsigned char *buffer = new unsigned char[size * size];
	RotateArray(buffer, data + size * size * type, size, angle);
	memcpy(data + (size * size * type), buffer, sizeof(unsigned char) * size * size);
	delete[] buffer;
	// ����P���۾F����L�����C
	unsigned char *buffers[4];
	// �N��ӭ������s��buffers��
	for (int i = 0; i < 4; i++) {
		int dst = (i + angle) % 4;
		buffers[dst] = new unsigned char[size * size];
		RotateArray(buffers[dst], data + size * size * relatedFace[type][i], size, relatedFaceDirection[type][dst] - relatedFaceDirection[type][i]);
	}
	// �̾�column�N�A�q����Ʀs�^data��
	for (int i = 0; i < 4; i++) {
		if (relatedFaceDirection[type][i] == 0) {
			memcpy(data + size * size * relatedFace[type][i], buffers[i], sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 1) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y + (size - column), buffers[i] + size * y + (size - column), sizeof(unsigned char) * column);
			}
		} else if (relatedFaceDirection[type][i] == 2) {
			memcpy(data + size * size * relatedFace[type][i] + size * (size - column), buffers[i] + size * (size - column), sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 3) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y, buffers[i] + size * y, sizeof(unsigned char) * column);
			}
		}
		delete[] buffers[i];
	}
}

void RubiksCube::Redo() {
	// TODO
}

void RubiksCube::Undo() {
	// TODO
}

std::ostream& operator<<(std::ostream& outputStream, const RubiksCube& cube) {
	static const char color[] = { 'G', 'R', 'W', 'Y', 'O', 'B' };
	std::string padding(cube.size * 2, ' ');
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 2 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 4 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 0 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 1 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 5 + cube.size * i + j]] << ' ';
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 3 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	outputStream << "htm: " << cube.htm << ", qtm: " << cube.qtm << std::endl;
	return outputStream;
}
