#include "RubiksCube.h"
#include <string>
#include <iostream>

RubiksCube::RubiksCube(int size) {
	this->size = size;
	this->data = new unsigned char[size * size * 6];
	Reset();
}

RubiksCube::~RubiksCube() {
	delete[] this->data;
}

void RubiksCube::Reset() {
	this->htm = 0;
	this->qtm = 0;
	this->lastFace = -1;
	memset(this->data + size * size * 0, 0, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 1, 1, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 2, 2, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 3, 3, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 4, 4, sizeof(unsigned char) * size * size);
	memset(this->data + size * size * 5, 5, sizeof(unsigned char) * size * size);
}

void RubiksCube::Shuffle() {
	for (size_t i = 0; i < 200; i++)
	{
		int type = rand() % 6;
		int column = (rand() % (size - 1)) + 1;
		int angle = (rand() % 3) + 1;
		Rotate(type, column, angle);
	}
}

void RubiksCube::Solve() {
	if (SolveTree(1, 15)) {
		std::cout << "success" << std::endl;
	}
	else {
		std::cout << "fail" << std::endl;
	}
}

bool RubiksCube::SolveTree(int turn, int maxTurn)
{
	if (turn > maxTurn)
		return false;
	for (size_t type = 0; type < 6; type++)
	{
		for (size_t column = 1; column < size; column++)
		{
			for (size_t angle = 1; angle < 4; angle++)
			{
				Rotate(type, column, angle);
				if (isSolved())
					return true;
				else if (SolveTree(turn + 1, maxTurn))
					return true;
				else
					Rotate(type, column, 4 - angle);
			}
		}
	}
	return false;
}

void RubiksCube::Rotate(OperateType type, int column, int angle) {
	Rotate((int)type, angle, column);
}

// ��x�}����M��g��t�@�ӯx�}
void RotateArray(unsigned char *dst, unsigned char *src, int size, int angle) {
	angle = (angle % 4 + 4) % 4;
	for (int y = 0; y < size; y++) {
		for (int x = 0; x < size; x++) {
			int index = y * size + x;
			int newIndex = 0;
			if (angle == 0) newIndex = y * size + x;
			if (angle == 1) newIndex = x * size + (size - 1 - y);
			if (angle == 2) newIndex = (size - 1 - y) * size + (size - 1 - x);
			if (angle == 3) newIndex = (size - 1 - x) * size + y;
			dst[newIndex] = src[index];
		}
	}
}

// �����C�ӭ��P�򦳭��X�ӭ�(���ɰw�A�W�k�U��)
static const int relatedFace[6][4] = {
	{ 2, 1, 5, 4 },
	{ 2, 3, 5, 0 },
	{ 3, 1, 0, 4 },
	{ 2, 4, 5, 1 },
	{ 2, 0, 5, 3 },
	{ 0, 1, 3, 4 },
};

// �����C�ӭ��P�򪺭��P�ۤv�۳s����V
static const int relatedFaceDirection[6][4] = {
	{ 2, 3, 0, 1 },
	{ 1, 3, 1, 1 },
	{ 0, 0, 0, 0 },
	{ 0, 3, 2, 1 },
	{ 3, 3, 3, 1 },
	{ 2, 2, 2, 2 },
};

void RubiksCube::Rotate(int type, int column, int angle) {
	htm += lastFace != type;
	lastFace = type;
	qtm += (angle == 2) + 1;
	// �����ӭ�
	unsigned char *buffer = new unsigned char[size * size];
	RotateArray(buffer, data + size * size * type, size, angle);
	memcpy(data + (size * size * type), buffer, sizeof(unsigned char) * size * size);
	delete[] buffer;
	// ����P���۾F����L�����C
	unsigned char *buffers[4];
	// �N��ӭ������s��buffers��
	for (int i = 0; i < 4; i++) {
		int dst = (i + angle) % 4;
		buffers[dst] = new unsigned char[size * size];
		RotateArray(buffers[dst], data + size * size * relatedFace[type][i], size, relatedFaceDirection[type][dst] - relatedFaceDirection[type][i]);
	}
	// �̾�column�N�A�q����Ʀs�^data��
	for (int i = 0; i < 4; i++) {
		if (relatedFaceDirection[type][i] == 0) {
			memcpy(data + size * size * relatedFace[type][i], buffers[i], sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 1) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y + (size - column), buffers[i] + size * y + (size - column), sizeof(unsigned char) * column);
			}
		} else if (relatedFaceDirection[type][i] == 2) {
			memcpy(data + size * size * relatedFace[type][i] + size * (size - column), buffers[i] + size * (size - column), sizeof(unsigned char) * size * column);
		} else if (relatedFaceDirection[type][i] == 3) {
			for (int y = 0; y < size; y++) {
				memcpy(data + size * size * relatedFace[type][i] + size * y, buffers[i] + size * y, sizeof(unsigned char) * column);
			}
		}
		delete[] buffers[i];
	}
}

void RubiksCube::Redo() {
	// TODO
}

void RubiksCube::Undo() {
	// TODO
}

bool RubiksCube::isSolved()
{
	for (size_t i = 0; i < size; i++)
	{
		for (size_t j = 0; j < size; j++)
		{
			if (data[size*size * 0 + size*i + j] != 0)return false;
			if (data[size*size * 1 + size*i + j] != 1)return false;
			if (data[size*size * 2 + size*i + j] != 2)return false;
			if (data[size*size * 3 + size*i + j] != 3)return false;
			if (data[size*size * 4 + size*i + j] != 4)return false;
			if (data[size*size * 5 + size*i + j] != 5)return false;
		}
	}
	return true;
}

bool RubiksCube::checkMe()
{
	int t[100];
	int a[100];
	int c[100];
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		t[i] = rand() % 6;
		a[i] = (rand() % 3) + 1;
		c[i] = (rand() % (size - 1)) + 1;
		//	std::cout << cube;
	}
	for (size_t i = 0; i < 100; i++)
	{
		Rotate(t[i], c[i], a[i]);
	}
	std::cout << *this;
	for (size_t i = 0; i < 100; i++)
	{
		int index = 99 - i;
		Rotate(t[index], c[index], 4 - a[index]);
	}
	std::cout << *this;
	return false;
}

std::ostream& operator<<(std::ostream& outputStream, const RubiksCube& cube) {
	static const char color[] = { 'G', 'R', 'W', 'B', 'O', 'Y' };
	std::string padding(cube.size * 2, ' ');
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 2 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 4 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 0 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 1 + cube.size * i + j]] << ' ';
		for (int j = 0; j < cube.size; j++) outputStream << color[cube.data[cube.size * cube.size * 3 + cube.size * i + j]] << ' ';
		outputStream << std::endl;
	}
	for (int i = 0; i < cube.size; i++) {
		outputStream << padding;
		for (int j = 0; j < cube.size; j++) {
			outputStream << color[cube.data[cube.size * cube.size * 5 + cube.size * i + j]] << ' ';
		}
		outputStream << std::endl;
	}
	outputStream << "htm: " << cube.htm << ", qtm: " << cube.qtm << std::endl;
	return outputStream;
}
